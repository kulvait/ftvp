#include "memory-color.cuh"
#include <stdlib.h>

// Handle errors raised by the GPU
// From
// http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                                             \
    {                                                                                              \
        gpuAssert((ans), __FILE__, __LINE__);                                                      \
    }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if(code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if(abort)
            exit(code);
    }
}

// Memory management
extern void init_cuda()
{
    double* dummy;
    gpuErrchk(hipMalloc((void**)&dummy, sizeof(double)));
    gpuErrchk(hipFree(dummy));
}

extern void
init_memory(struct memory_cuda* mem, struct res_cuda* res, int sx, int sy, int sc, double* u)
{
    // GPU buffers
    double *dev_xie, *dev_xio,
        *dev_u; // gpu,  xie -> even coordinates of xi, xio -> odd coordinates
    double* dev_xiobar; // gpu, xiobar -> over-relaxation of xio
    double *dev_gle, *dev_glo; // gap arrays

    res->it = 0;
    res->msec = 0;
    res->gap = 0;
    res->rmse = 0;

    int sxyz = sx * sy * sc;
    int Ke = sx / 2, Le = sy / 2;
    int Ko = (sx - 2) / 2, Lo = (sy - 2) / 2;

    int factor = 4;

    // Memory management
    gpuErrchk(hipMalloc((void**)&dev_xie, Ke * Le * sc * factor * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&dev_xio, Ko * Lo * sc * factor * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&dev_xiobar, Ko * Lo * sc * factor * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&dev_u, sxyz * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&dev_gle, Ke * Le * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&dev_glo, Ko * Lo * sizeof(double)));

    gpuErrchk(hipMemcpy(dev_u, u, sxyz * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(dev_xie, 0, Ke * Le * sc * factor * sizeof(double)));
    gpuErrchk(hipMemset(dev_xio, 0, Ko * Lo * sc * factor * sizeof(double)));
    // Additional sanitization, not in original code
    gpuErrchk(hipMemset(dev_xiobar, 0, Ko * Lo * sc * factor * sizeof(double)));
    gpuErrchk(hipMemset(dev_gle, 0, Ke * Le * sizeof(double)));
    gpuErrchk(hipMemset(dev_glo, 0, Ko * Lo * sizeof(double)));

    mem->dev_xie = dev_xie;
    mem->dev_xio = dev_xio;
    mem->dev_u = dev_u;
    mem->dev_xiobar = dev_xiobar;
    mem->dev_gle = dev_gle;
    mem->dev_glo = dev_glo;
    return;
}

extern void free_memory(struct memory_cuda* mem)
{
    gpuErrchk(hipFree(mem->dev_xie));
    gpuErrchk(hipFree(mem->dev_xio));
    gpuErrchk(hipFree(mem->dev_xiobar));
    gpuErrchk(hipFree(mem->dev_u));
    gpuErrchk(hipFree(mem->dev_gle));
    gpuErrchk(hipFree(mem->dev_glo));
    return;
}

