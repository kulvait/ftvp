#include <stdlib.h>
#include "memory.cuh"

// Handle errors raised by the GPU
// From http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Memory management
extern
void init_cuda() {
  double *dummy;
  gpuErrchk( hipMalloc((void**) &dummy, sizeof(double)) );
  gpuErrchk( hipFree(dummy) );
}

extern
void init_memory(struct memory_cuda *mem, struct res_cuda *res, int sx, int sy, int sz, double* u) {
  double *xie, *xio; // xie -> even coordinates of xi, xio -> odd coordinates
  double *xiobar; // xiobar -> over-relaxation of xio
  double *gle, *glo; // gap arrays

  // GPU versions
  double *dev_xie, *dev_xio, *dev_u; // gpu versions
  double *dev_xiobar, *dev_xioswp; // gpu versions
  double *dev_gle, *dev_glo; // gap array

  res->it = 0;
  res->msec = 0;
  res->gap = 0;
  res->rmse = 0;

  int sxyz = sx * sy * sz;
  int Ke = sx/2, Le=sy/2;
  int Me = sz > 1 ? sz/2 : 1;
  int Ko = (sx-2)/2, Lo = (sy-2)/2;
  int Mo = sz > 1 ? (sz-2) : 1;

  int factor = sz > 1 ? 12 : 4;

  // Memory management
  xie = (double *) malloc(Ke*Le*Me*factor*sizeof(double));
  xio = (double *) malloc(Ko*Lo*Mo*factor*sizeof(double));
  xiobar = (double *) malloc(Ko*Lo*Mo*factor*sizeof(double));
  gle = (double *) malloc(Ke*Le*Me*sizeof(double));
  glo = (double *) malloc(Ko*Lo*Mo*sizeof(double));
  memset(xie, 0, Ke*Le*Me*factor*sizeof(double));
  memset(xio, 0, Ko*Lo*Mo*factor*sizeof(double));

  gpuErrchk( hipMalloc((void**)&dev_xie, Ke*Le*Me*factor*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&dev_xio, Ko*Lo*Mo*factor*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&dev_xiobar, Ko*Lo*Mo*factor*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&dev_xioswp, Ko*Lo*Mo*factor*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&dev_u, sxyz*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&dev_gle, Ke*Le*Me*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&dev_glo, Ko*Lo*Mo*sizeof(double)) );

  gpuErrchk( hipMemcpy(dev_xie, xie, Ke*Le*Me*factor*sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(dev_xio, xio, Ko*Lo*Mo*factor*sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(dev_xiobar, xiobar, Ko*Lo*Mo*factor*sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(dev_u, u, sxyz*sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(dev_gle, gle, Ke*Le*Me*sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(dev_glo, glo, Ko*Lo*Mo*sizeof(double), hipMemcpyHostToDevice) );

  mem->dev_xie = dev_xie;
  mem->dev_xio = dev_xio;
  mem->dev_u = dev_u;
  mem->dev_xiobar = dev_xiobar;
  mem->dev_xioswp = dev_xioswp;
  mem->dev_gle = dev_gle;
  mem->dev_glo = dev_glo;
  return;
}

extern
void free_memory(struct memory_cuda *mem) {
  gpuErrchk( hipFree(mem->dev_xie) );
  gpuErrchk( hipFree(mem->dev_xio) );
  gpuErrchk( hipFree(mem->dev_xiobar) );
  gpuErrchk( hipFree(mem->dev_u) );
  gpuErrchk( hipFree(mem->dev_gle) );
  gpuErrchk( hipFree(mem->dev_glo) );
  free(mem);
  return;
}

