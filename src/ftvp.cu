#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "ftvp.cuh"
#include "kernels/kernels-2d.cuh"
#include "memory.cuh"

// Handle errors raised by the GPU
// From
// http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                                             \
    {                                                                                              \
        gpuAssert((ans), __FILE__, __LINE__);                                                      \
    }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if(code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if(abort)
            exit(code);
    }
}

// Return the exec time in ms.
inline double bench_time(timeval start, timeval end)
{
    return (end.tv_sec * 1000.0 + (end.tv_usec / 1000.0))
        - (start.tv_sec * 1000.0 + (start.tv_usec / 1000.0));
}

extern void prox_tv_2d_noalloc(struct memory_cuda* mem,
                               struct res_cuda* res,
                               int sx,
                               int sy,
                               double lambda,
                               double epsilon,
                               int Nit,
                               int block_size,
                               int steps,
                               int gapiter,
                               double gap_factor,
                               int use_newton,
                               int upd_strat)
{
    int it = 0; // current iteration
    double t = 1; // relaxation parameter
    double tt;
    double gamma, gammap;
    double theta = 0; // relaxation parameter
    double gap = 0;
    timeval start, end;

    int sxy = sx * sy;
    double w = 2 * lambda * lambda;
    double ws = M_SQRT2 * lambda; // sqrt(w)
    epsilon /= ws; //  |Du|_C threshold
    double we2 = w * epsilon * epsilon;

    double tau = 0.25;
    double q;
    if(upd_strat == UPDATE_VARYING)
    {
        gamma = epsilon * tau;
        gammap = gamma + gamma / (1 + gamma);
        q = gammap / (1 + gamma);
    } else
    {
        q = tau * epsilon / (1 + tau * epsilon);
    }

    int Ke = sx / 2, Le = sy / 2;
    int Ko = (sx - 2) / 2, Lo = (sy - 2) / 2;

    dim3 blocks_odd((Ko + block_size - 1) / block_size, (Lo + block_size - 1) / block_size);
    dim3 blocks_even((Ke + block_size - 1) / block_size, (Le + block_size - 1) / block_size);
    dim3 threads(block_size, block_size);

    gettimeofday(&start, NULL);
    double* dev_xioswp = nullptr;
    do
    {
        // Over-relaxation
        if(upd_strat == UPDATE_CONSTANT)
        {
            theta = 1 / (1 + tau * sqrt(epsilon));
            theta *= theta;
        } else if(upd_strat == UPDATE_VARYING)
        {
            tt = t;
            t = .5
                * ((1 - q * t * t)
                   + sqrt((1 - q * t * t) * (1 - q * t * t)
                          + 4 * (1 + gammap) / (1 + gamma) * t * t));
            theta = (tt - 1) / t * (1 - (t - 1) * gammap);
        } else
        {
            tt = t;
            t = .5 * ((1. - q * t * t) + sqrt((1 - q * t * t) * (1 - q * t * t) + 4 * t * t));
            theta = (tt - 1) / t * (1 - (t - 1) * tau * epsilon);
        }
        if(theta > 0)
        {
            over_relax_eps_2d<<<blocks_odd, threads>>>(sx, mem->dev_xio, mem->dev_xiobar,
                                                       mem->dev_u, theta, Lo, Ko, 0);
            // Swap xio and xiobar
            dev_xioswp = mem->dev_xio;
            mem->dev_xio = mem->dev_xiobar;
            mem->dev_xiobar = dev_xioswp;
        }

        // Minimization
        opt_eps_split<<<blocks_even, threads>>>(sx, mem->dev_xie, mem->dev_u, epsilon, w, ws, we2,
                                                Le, Ke, steps, use_newton, 1);
        opt_eps_split<<<blocks_odd, threads>>>(sx, mem->dev_xiobar, mem->dev_u, epsilon, w, ws, we2,
                                               Lo, Ko, steps, use_newton, 0);

        // Gap
        if(it % gapiter == 0)
        {
            gap_arr_eps_2d<<<blocks_odd, threads>>>(sx, mem->dev_glo, mem->dev_xio, mem->dev_u,
                                                    epsilon, w, ws, we2, Lo, Ko, 0);
            thrust::device_ptr<double> D = thrust::device_pointer_cast(mem->dev_glo);
            gap = thrust::reduce(D, D + Ko * Lo, (double)0, thrust::plus<double>());
            gap_arr_eps_2d<<<blocks_even, threads>>>(sx, mem->dev_gle, mem->dev_xie, mem->dev_u,
                                                     epsilon, w, ws, we2, Le, Ke, 1);
            D = thrust::device_pointer_cast(mem->dev_gle);
            gap += thrust::reduce(D, D + Ke * Le, (double)0, thrust::plus<double>());
        }
        it++;
    } while(it < Nit && gap > sxy * gap_factor);
    gettimeofday(&end, NULL);

    double msec = bench_time(start, end);

    res->it = it;
    res->msec = msec;
    res->gap = gap;
    res->rmse = sqrt(gap / (sx * sy));
    return;
}

extern res_cuda* prox_tv(int sx,
                         int sy,
                         int sz,
                         double* u,
                         double lambda,
                         double epsilon,
                         int Nit,
                         int block_size,
                         int steps,
                         int gapiter,
                         double gap_factor,
                         OptimizationMethod opt_meth,
                         int upd_strat)
{
    struct res_cuda* res = (struct res_cuda*)malloc(sizeof(struct res_cuda));
    struct memory_cuda* mem = (struct memory_cuda*)malloc(sizeof(struct memory_cuda));
    int use_newton = opt_meth == OE_SPLIT_NEWTON ? 1 : 0;

    init_memory(mem, res, sx, sy, sz, u);
    if(sz > 1)
    {
        exit(1);
    } else
    {
        prox_tv_2d_noalloc(mem, res, sx, sy, lambda, epsilon, Nit, block_size, steps, gapiter,
                           gap_factor, use_newton, upd_strat);
    }

    // Memory management
    gpuErrchk(hipMemcpy(u, mem->dev_u, sx * sy * sz * sizeof(double), hipMemcpyDeviceToHost));
    free_memory(mem);
    return res;
}
